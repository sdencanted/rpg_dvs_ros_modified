#include "hip/hip_runtime.h"
#include "dvxplorer_mc_ceres/motion_compensation.cuh"

#include <cmath>
#include <algorithm> //for std::max
#include <cstdio>
#include <vector>
#include <iostream>
#include "dvxplorer_mc_ceres/utils.h"

#include <hipcub/hipcub.hpp>
#include <cub/block/block_reduce.cuh>

#include <jetson-utils/cudaMappedMemory.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_cooperative_groups.h>

#define FULL_MASK 0xffffffff

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template <class T>
struct SharedMemory
{
    __device__ inline operator T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

__global__ void g_motionCompensate(float fx, float fy, float cx, float cy, int height, int width, int num_events, const float *x_unprojected, const float *y_unprojected, float *x_prime, float *y_prime, float *t, const float rotation_x, const float rotation_y, const float rotation_z, int x_offset, int y_offset)
{
    // size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    int total_cx = cx - x_offset;
    int total_cy = cy - y_offset;
    // if (i < num_events)
    for (size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x); i < num_events; i += num_threads_in_grid)
    {
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t[i];
        float theta_y_t = rotation_y * t[i];
        float theta_z_t = rotation_z * t[i];

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + total_cx;
        y_prime[i] = fy * y_rotated_norm + total_cy;
    }
}

__global__ void g_motionCompensateAndFillImageGaussian5(float fx, float fy, float cx, float cy, int height, int width, int num_events, const float *x_unprojected, const float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int x_offset, int y_offset)
{

    float image_sum = 0;
    float image_sum_del_theta_x = 0;
    float image_sum_del_theta_y = 0;
    float image_sum_del_theta_z = 0;
    float *image_del_x = image + height * width;
    float *image_del_y = image + height * width * 2;
    float *image_del_z = image + height * width * 3;
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    float t_mid = (t[num_events - 1] + t[0]) / 2;
    int total_cx = cx - x_offset;
    int total_cy = cy - y_offset;
    for (size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x); i < num_events; i += num_threads_in_grid)
    {
        float t_norm = t[i] - t_mid;
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t_norm;
        float theta_y_t = rotation_y * t_norm;
        float theta_z_t = rotation_z * t_norm;

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + total_cx;
        y_prime[i] = fy * y_rotated_norm + total_cy;
        // populate image
        int x_round = round(x_prime[i]);
        int y_round = round(y_prime[i]);
        float gaussian;

        if (x_round >= 1 && x_round <= width && y_round >= 1 && y_round <= height)
        {
            float fx_div_z_rotated_ti = fx * z_rotated_inv * t_norm;
            float fy_div_z_rotated_ti = fy * z_rotated_inv * t_norm;
            float del_x_del_theta_y = fx_div_z_rotated_ti * (1 + x_unprojected[i] * x_rotated_norm);
            float del_x_del_theta_z = -fx_div_z_rotated_ti * y_unprojected[i];
            float del_x_del_theta_x = del_x_del_theta_z * x_rotated_norm;
            float del_y_del_theta_x = fy_div_z_rotated_ti * (-1 - y_unprojected[i] * y_rotated_norm);
            float del_y_del_theta_z = fy_div_z_rotated_ti * x_unprojected[i];
            float del_y_del_theta_y = del_y_del_theta_z * y_rotated_norm;

            for (int row = max(1, y_round - 2); row < min(height, y_round + 3); row++)
            {
                for (int col = max(1, x_round - 2); col < min(width, x_round + 3); col++)
                {
                    float x_diff = col - x_prime[i];
                    float y_diff = row - y_prime[i];
                    gaussian = exp((-x_diff * x_diff - y_diff * y_diff) / 2);
                    int idx = (row - 1) * (width) + col - 1;
                    atomicAdd(&image[idx], gaussian);
                    image_sum += gaussian;
                    float del_x = gaussian * (x_diff * del_x_del_theta_x + y_diff * del_y_del_theta_x);
                    atomicAdd(&image_del_x[idx], del_x);
                    image_sum_del_theta_x += del_x;
                    float del_y = gaussian * (x_diff * del_x_del_theta_y + y_diff * del_y_del_theta_y);
                    atomicAdd(&image_del_y[idx], del_y);
                    image_sum_del_theta_y += del_y;
                    float del_z = gaussian * (x_diff * del_x_del_theta_z + y_diff * del_y_del_theta_z);
                    atomicAdd(&image_del_z[idx], del_z);
                    image_sum_del_theta_z += del_z;
                }
            }
        }
    }
    float *sdata = SharedMemory<float>();
    uint16_t tid = threadIdx.x;

    // do reduction in shared mem

    // sum up to 128 elements

    float temp_sum;
    // image_sum
    sdata[tid] = image_sum;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum = image_sum + sdata[tid + 256];
    __syncthreads();
    // store contrast in 0 to 127
    if (tid < 128)
        temp_sum = image_sum + sdata[tid + 128];
    __syncthreads();
    // image_sum_del_theta_x
    sdata[tid] = image_sum_del_theta_x;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 128];
    __syncthreads();
    // store x in 128 to 255
    if (tid >= 128 && tid < 256)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_sum_del_theta_y
    sdata[tid] = image_sum_del_theta_y;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 128];
    __syncthreads();
    // store y in 256 to 383
    if (tid >= 256 && tid < 384)
    {
        temp_sum = sdata[tid - 256];
    }
    __syncthreads();
    // image_sum_del_theta_z
    sdata[tid] = image_sum_del_theta_z;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
    {
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 128];
    }
    __syncthreads();
    // store z in 384 to 512
    if (tid >= 384)
    {
        temp_sum = sdata[tid - 384];
    }
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x7F) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid & 0x7F) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    __syncthreads();

    if (tid == 0)
    {
        // image_sum
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_sum_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 256)
    {
        // image_sum_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 384)
    {
        // image_sum_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}

__global__ void g_motionCompensateAndFillImageBilinear(float fx, float fy, float cx, float cy, int height, int width, int num_events, const float *x_unprojected, const float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int x_offset, int y_offset)
{

    float image_sum = 0;
    float image_sum_del_theta_x = 0;
    float image_sum_del_theta_y = 0;
    float image_sum_del_theta_z = 0;
    float *image_del_x = image + height * width;
    float *image_del_y = image + height * width * 2;
    float *image_del_z = image + height * width * 3;
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    float t_mid = (t[num_events - 1] + t[0]) / 2;
    int total_cx = cx - x_offset;
    int total_cy = cy - y_offset;
    for (size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x); i < num_events; i += num_threads_in_grid)
    {
        float t_norm = t[i] - t_mid;
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t_norm;
        float theta_y_t = rotation_y * t_norm;
        float theta_z_t = rotation_z * t_norm;

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + total_cx;
        y_prime[i] = fy * y_rotated_norm + total_cy;
        // populate image

        // Bilinear
        int x_trunc = int(x_prime[i]);
        int y_trunc = int(y_prime[i]);
        if (x_trunc >= 1 && x_trunc <= width - 2 && y_trunc >= 1 && y_trunc <= height - 2)
        {

            // int idx1 = x_trunc - 1 + (y_trunc - 1) * width;
            // int idx2 = idx1 + 1;
            // int idx3 = idx1 + width;
            // int idx4 = idx3 + 1;

            int idx4 = x_trunc + y_trunc * width;
            int idx3 = idx4 - 1;
            int idx2 = idx4 - width;
            int idx1 = idx2 - 1;
            float x_diff = x_prime[i] - x_trunc;
            float y_diff = y_prime[i] - y_trunc;
            float del_x_del_theta_x, del_x_del_theta_y, del_x_del_theta_z, del_y_del_theta_x, del_y_del_theta_y, del_y_del_theta_z;
            float fx_div_z_rotated_ti = fx * z_rotated_inv * t[i];
            float fy_div_z_rotated_ti = fy * z_rotated_inv * t[i];
            del_x_del_theta_y = fx_div_z_rotated_ti * (1 + x_unprojected[i] * x_rotated_norm);
            del_x_del_theta_z = fx_div_z_rotated_ti * -y_unprojected[i];
            del_x_del_theta_x = del_x_del_theta_z * x_rotated_norm;
            del_y_del_theta_x = fy_div_z_rotated_ti * (-1 - y_unprojected[i] * y_rotated_norm);
            del_y_del_theta_z = fy_div_z_rotated_ti * x_unprojected[i];
            del_y_del_theta_y = del_y_del_theta_z * y_rotated_norm;
            // float d1x = -(1 - y_diff);
            // float d1y = -(1 - x_diff);
            float d2x = 1 - y_diff;
            float d2y = -x_diff;
            float d3x = -y_diff;
            float d3y = 1 - x_diff;
            float d4x = y_diff;
            float d4y = x_diff;

            float d1x = -d2x;
            float d1y = -d3y;

            float im1 = d3y * d2x;
            float im2 = d4y * d2x;
            float im3 = d3y * y_diff;
            float im4 = (x_diff) * (y_diff);
            image_sum += im1 + im2 + im3 + im4;
            atomicAdd(&image[idx1], im1);
            atomicAdd(&image[idx2], im2);
            atomicAdd(&image[idx3], im3);
            atomicAdd(&image[idx4], im4);
            float dx1 = d1x * del_x_del_theta_x + d1y * del_y_del_theta_x;
            float dx2 = d2x * del_x_del_theta_x + d2y * del_y_del_theta_x;
            float dx3 = d3x * del_x_del_theta_x + d3y * del_y_del_theta_x;
            float dx4 = d4x * del_x_del_theta_x + d4y * del_y_del_theta_x;
            image_sum_del_theta_x += dx1 + dx2 + dx3 + dx4;

            atomicAdd(&image_del_x[idx1], dx1);
            atomicAdd(&image_del_x[idx2], dx2);
            atomicAdd(&image_del_x[idx3], dx3);
            atomicAdd(&image_del_x[idx4], dx4);
            float dy1 = d1x * del_x_del_theta_y + d1y * del_y_del_theta_y;
            float dy2 = d2x * del_x_del_theta_y + d2y * del_y_del_theta_y;
            float dy3 = d3x * del_x_del_theta_y + d3y * del_y_del_theta_y;
            float dy4 = d4x * del_x_del_theta_y + d4y * del_y_del_theta_y;
            image_sum_del_theta_y += dy1 + dy2 + dy3 + dy4;
            atomicAdd(&image_del_y[idx1], dy1);
            atomicAdd(&image_del_y[idx2], dy2);
            atomicAdd(&image_del_y[idx3], dy3);
            atomicAdd(&image_del_y[idx4], dy4);
            float dz1 = d1x * del_x_del_theta_z + d1y * del_y_del_theta_z;
            float dz2 = d2x * del_x_del_theta_z + d2y * del_y_del_theta_z;
            float dz3 = d3x * del_x_del_theta_z + d3y * del_y_del_theta_z;
            float dz4 = d4x * del_x_del_theta_z + d4y * del_y_del_theta_z;
            image_sum_del_theta_z += dz1 + dz2 + dz3 + dz4;
            atomicAdd(&image_del_z[idx1], dz1);
            atomicAdd(&image_del_z[idx2], dz2);
            atomicAdd(&image_del_z[idx3], dz3);
            atomicAdd(&image_del_z[idx4], dz4);
        }
    }
    float *sdata = SharedMemory<float>();
    uint16_t tid = threadIdx.x;

    // do reduction in shared mem

    // sum up to 128 elements

    float temp_sum;
    // image_sum
    sdata[tid] = image_sum;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum = image_sum + sdata[tid + 256];
    __syncthreads();
    // store contrast in 0 to 127
    if (tid < 128)
        temp_sum = image_sum + sdata[tid + 128];
    __syncthreads();
    // image_sum_del_theta_x
    sdata[tid] = image_sum_del_theta_x;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 128];
    __syncthreads();
    // store x in 128 to 255
    if (tid >= 128 && tid < 256)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_sum_del_theta_y
    sdata[tid] = image_sum_del_theta_y;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 128];
    __syncthreads();
    // store y in 256 to 383
    if (tid >= 256 && tid < 384)
    {
        temp_sum = sdata[tid - 256];
    }
    __syncthreads();
    // image_sum_del_theta_z
    sdata[tid] = image_sum_del_theta_z;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
    {
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 128];
    }
    __syncthreads();
    // store z in 384 to 512
    if (tid >= 384)
    {
        temp_sum = sdata[tid - 384];
    }
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x7F) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid & 0x7F) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    __syncthreads();

    if (tid == 0)
    {
        // image_sum
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_sum_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 256)
    {
        // image_sum_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 384)
    {
        // image_sum_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}

__global__ void g_fillImageKroneckerNoJacobians(int height, int width, int num_events, float *x_prime, float *y_prime, int *image)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // populate image
        // check if coordinates are 3 pixels in of the boundary
        int x_round = round(x_prime[i]);
        int y_round = round(y_prime[i]);
        if (x_round >= 1 && x_round <= width && y_round >= 1 && y_round <= height)
        {
            int idx = (y_round - 1) * width + x_round - 1;
            atomicAdd(&image[idx], 1);
        }
    }
}
__global__ void g_fillImageBilinearNoJacobians(int height, int width, int num_events, float *x_prime, float *y_prime, float *image)
{
    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // Bilinear
        int x_trunc = int(x_prime[i]);
        int y_trunc = int(y_prime[i]);
        if (x_trunc >= 1 && x_trunc <= width - 2 && y_trunc >= 1 && y_trunc <= height - 2)
        {

            int idx4 = x_trunc + y_trunc * width;
            int idx3 = idx4 - 1;
            int idx2 = idx4 - width;
            int idx1 = idx2 - 1;
            float x_diff = x_prime[i] - x_trunc;
            float y_diff = y_prime[i] - y_trunc;
            float d2x = 1 - y_diff;
            float d3y = 1 - x_diff;
            float d4y = x_diff;

            float im1 = d3y * d2x;
            float im2 = d4y * d2x;
            float im3 = d3y * y_diff;
            float im4 = (x_diff) * (y_diff);
            atomicAdd(&image[idx1], im1);
            atomicAdd(&image[idx2], im2);
            atomicAdd(&image[idx3], im3);
            atomicAdd(&image[idx4], im4);
        }
    }
}
__device__ volatile float mean_volatile[4] = {0};

template <int block_size>
__global__ void g_calculateAndReduceContrastAndJacobians(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize)
{
    cooperative_groups::grid_group grid = cooperative_groups::this_grid();
    float *image_del_x = image + num_elements;
    float *image_del_y = image + num_elements * 2;
    float *image_del_z = image + num_elements * 3;
    // START COPY
    float *sdata = SharedMemory<float>();
    float temp_sum = 0;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    // if (tid < prev_gridsize)
    for (int i = tid; i < prev_gridsize; i += blockDim.x)
    {

        if (blockIdx.x == 0)
        {
            temp_sum += contrast_block_sum[i];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum += contrast_del_x_block_sum[i];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum += contrast_del_y_block_sum[i];
        }
        else if (blockIdx.x == 3)
        {
            temp_sum += contrast_del_z_block_sum[i];
        }
    }
    sdata[tid] = temp_sum;
    __syncthreads();
    if (block_size > 256)
    {
        // 512 threads per block
        if ((tid) < 256)
        {
            sdata[tid] = temp_sum = temp_sum + sdata[tid + 128];
        }
        __syncthreads();
    }
    if (block_size > 128)
    {
        if ((tid) < 128)
        {
            sdata[tid] = temp_sum = temp_sum + sdata[tid + 128];
        }
        __syncthreads();
    }
    if (block_size > 64)
    {
        if ((tid) < 64)
        {
            sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
        }
        __syncthreads();
    }
    if ((tid) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        // write means
        if (blockIdx.x < 4)
        {
            mean_volatile[blockIdx.x] = temp_sum / num_elements;
        }
    }
    // __syncthreads();

    // END COPY
    float image_contrast = 0;
    float image_contrast_del_theta_x = 0;
    float image_contrast_del_theta_y = 0;
    float image_contrast_del_theta_z = 0;
    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t idx = thread_grid_idx;
    cooperative_groups::sync(grid);
    float mean = mean_volatile[0];
    float xmean = mean_volatile[1];
    float ymean = mean_volatile[2];
    float zmean = mean_volatile[3];

    while (idx < num_elements)
    {
        float image_norm = image[idx] - mean;
        float image_norm_x = image_del_x[idx] - xmean;
        float image_norm_y = image_del_y[idx] - ymean;
        float image_norm_z = image_del_z[idx] - zmean;
        image_contrast += image_norm * image_norm;
        image_contrast_del_theta_x += image_norm_x * image_norm;
        image_contrast_del_theta_y += image_norm_y * image_norm;
        image_contrast_del_theta_z += image_norm_z * image_norm;
        // imagedebug[idx] = image_norm * image_norm;
        // imagedebug[idx + num_elements] = image_norm_x * image_norm;
        // imagedebug[idx + num_elements * 2] = image_norm_y * image_norm;
        // imagedebug[idx + num_elements * 3] = image_norm_z * image_norm;
        idx += blockDim.x * gridDim.x;
    }
    // BEGIN DEBUG
    // sdata[tid] = 0;
    // if(blockIdx.x==0&&threadIdx.x==0){
    //     for(int idx_special=0;idx_special<num_elements;idx_special++)

    //     {
    //         float image_norm = image[idx_special] - mean;
    //         float image_norm_x = image_del_x[idx_special] - xmean;
    //         float image_norm_y = image_del_y[idx_special] - ymean;
    //         float image_norm_z = image_del_z[idx_special] - zmean;
    //         // image_contrast += image_norm * image_norm;
    //         // image_contrast_del_theta_x += image_norm_x * image_norm;
    //         // image_contrast_del_theta_y += image_norm_y * image_norm;
    //         // image_contrast_del_theta_z += image_norm_z * image_norm;
    //         // imagedebug[idx_special] = image_norm * image_norm;
    //         // imagedebug[idx_special + num_elements] = image_norm_x * image_norm;
    //         // imagedebug[idx_special + num_elements * 2] = image_norm_y * image_norm;
    //         // imagedebug[idx_special + num_elements * 3] = image_norm_z * image_norm;

    //         atomicAdd(&sdata[0], image_norm * image_norm);
    //         atomicAdd(&sdata[1], image_norm_x * image_norm);
    //         atomicAdd(&sdata[2], image_norm_y * image_norm);
    //         atomicAdd(&sdata[3], image_norm_z * image_norm);

    //     }

    //     contrast_block_sum[blockIdx.x] = sdata[0];
    //     contrast_del_x_block_sum[blockIdx.x] = sdata[1];
    //     contrast_del_y_block_sum[blockIdx.x] = sdata[2];
    //     contrast_del_z_block_sum[blockIdx.x] = sdata[3];
    //     // contrast_block_sum[blockIdx.x] = image_contrast;
    //     // contrast_del_x_block_sum[blockIdx.x] = image_contrast_del_theta_x;
    //     // contrast_del_y_block_sum[blockIdx.x] = image_contrast_del_theta_y;
    //     // contrast_del_z_block_sum[blockIdx.x] = image_contrast_del_theta_z;
    // }
    // else{

    //     contrast_block_sum[blockIdx.x] =0;
    //     contrast_del_x_block_sum[blockIdx.x] =0;
    //     contrast_del_y_block_sum[blockIdx.x] =0;
    //     contrast_del_z_block_sum[blockIdx.x] = 0;
    // }
    // return;
    // END DEBUG

    // do reduction in shared mem

    // sum up to 128 elements

    // float temp_sum;
    // image_contrast
    sdata[tid] = image_contrast;
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast = image_contrast + sdata[tid + 128];
    __syncthreads();
    // store contrast in 0 to 63
    if (tid < 64)
        temp_sum = image_contrast + sdata[tid + 64];
    __syncthreads();
    // image_contrast_del_theta_x
    sdata[tid] = image_contrast_del_theta_x;
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_x = image_contrast_del_theta_x + sdata[tid + 128];
    __syncthreads();
    if (tid < 64)
        sdata[tid] = image_contrast_del_theta_x = image_contrast_del_theta_x + sdata[tid + 64];
    __syncthreads();
    // store x in 64 to 127
    if (tid >= 64 && tid < 128)
    {
        temp_sum = sdata[tid - 64];
    }
    __syncthreads();
    // image_contrast_del_theta_y
    sdata[tid] = image_contrast_del_theta_y;
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_y = image_contrast_del_theta_y + sdata[tid + 128];
    __syncthreads();
    if (tid < 64)
        sdata[tid] = image_contrast_del_theta_y = image_contrast_del_theta_y + sdata[tid + 64];
    __syncthreads();
    // store y in 128 to 191
    if (tid >= 128 && tid < 192)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_contrast_del_theta_z
    sdata[tid] = image_contrast_del_theta_z;
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_z = image_contrast_del_theta_z + sdata[tid + 128];
    __syncthreads();
    if (tid < 64)
    {
        sdata[tid] = image_contrast_del_theta_z = image_contrast_del_theta_z + sdata[tid + 64];
    }
    __syncthreads();
    // store z in 192 to 255
    if (tid >= 192)
    {
        temp_sum = sdata[tid - 192];
    }
    __syncthreads();
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x3F) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }

    if (tid == 0)
    {
        // image_contrast
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 64)
    {
        // image_contrast_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_contrast_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 192)
    {
        // image_contrast_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}
template __global__ void g_calculateAndReduceContrastAndJacobians<512>(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize);
template __global__ void g_calculateAndReduceContrastAndJacobians<256>(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize);
template __global__ void g_calculateAndReduceContrastAndJacobians<128>(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize);
template __global__ void g_calculateAndReduceContrastAndJacobians<64>(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize);
// 4 blocks x threads
template <int prev_block_size>
__global__ void g_reduceContrastAndJacobiansPt2(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize)
{
    float *sdata = SharedMemory<float>();

    float temp_sum = 0;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    if (tid < prev_gridsize)
    {

        if (blockIdx.x == 0)
        {
            temp_sum = temp_sum = contrast_block_sum[tid];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum = contrast_del_x_block_sum[tid];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum = contrast_del_y_block_sum[tid];
        }
        else if (blockIdx.x == 3)
        {
            temp_sum = contrast_del_z_block_sum[tid];
        }
    }
    sdata[tid] = temp_sum;
    __syncthreads();

    if (prev_block_size > 256 && (tid) < 256)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 256];
    }
    __syncthreads();
    if (prev_block_size > 128 && (tid) < 128)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 128];
    }
    __syncthreads();
    if (prev_block_size > 64 && (tid) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        if (blockIdx.x == 0)
        {
            contrast_block_sum[0] = temp_sum;
        }
        else if (blockIdx.x == 1)
        {
            contrast_block_sum[1] = temp_sum;
        }
        else if (blockIdx.x == 2)
        {
            contrast_block_sum[2] = temp_sum;
        }
        else
        {
            contrast_block_sum[3] = temp_sum;
        }
    }
}
template __global__ void g_reduceContrastAndJacobiansPt2<512>(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize);
template __global__ void g_reduceContrastAndJacobiansPt2<256>(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize);
template __global__ void g_reduceContrastAndJacobiansPt2<128>(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize);
template __global__ void g_reduceContrastAndJacobiansPt2<64>(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize);

__global__ void g_rescaleIntensityInt(int *image, uint8_t *output_image, int maximum, int numel)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t idx = thread_grid_idx; idx < numel; idx += num_threads_in_grid)
    {
        output_image[idx] = (uint8_t)min(255, max(0, (int)(255 * image[idx] / (maximum / 2))));
    }
}
__global__ void g_rescaleIntensityFloat(float *image, uint8_t *output_image, float maximum, int numel)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t idx = thread_grid_idx; idx < numel; idx += num_threads_in_grid)
    {
        output_image[idx] = (uint8_t)min(255, max(0, (int)(255.0 * image[idx] / (maximum / 2))));
    }
}
