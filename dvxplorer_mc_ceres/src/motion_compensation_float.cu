#include "hip/hip_runtime.h"
#include "dvxplorer_mc_ceres/motion_compensation_float.h"

#include <cmath>
#include <algorithm> //for std::max
#include <cstdio>
#include <vector>
#include <iostream>
#include "dvxplorer_mc_ceres/utils.h"

#include <hipcub/hipcub.hpp>
#include <cub/block/block_reduce.cuh>

#include <jetson-utils/cudaMappedMemory.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_cooperative_groups.h>
#define FULL_MASK 0xffffffff
// using namespace cooperative_groups;
float thrustMean(float *image_, int height_, int width_)
{
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(image_);
    float sum1 = thrust::reduce(dev_ptr, dev_ptr + height_ * width_, 0.0, thrust::plus<float>());
    return sum1 / (height_ * width_);
}
float thrustSum(float *image_, int num_el)
{
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(image_);
    float sum1 = thrust::reduce(dev_ptr, dev_ptr + num_el, 0.0, thrust::plus<float>());
    return sum1;
}
// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template <class T>
struct SharedMemory
{
    __device__ inline operator T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

__global__ void fillImage_(float fx, float fy, float cx, float cy, int height, int width, int num_events, const float *x_unprojected, const float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int x_offset, int y_offset)
{

    float image_sum = 0;
    float image_sum_del_theta_x = 0;
    float image_sum_del_theta_y = 0;
    float image_sum_del_theta_z = 0;
    float *image_del_x = image + height * width;
    float *image_del_y = image + height * width * 2;
    float *image_del_z = image + height * width * 3;
    // size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    // if (i < num_events)
    float t_mid = (t[num_events - 1] + t[0]) / 2;
    for (size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x); i < num_events; i += num_threads_in_grid)
    {
        float t_norm = t[i] - t_mid;
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t_norm;
        float theta_y_t = rotation_y * t_norm;
        float theta_z_t = rotation_z * t_norm;

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + cx;
        y_prime[i] = fy * y_rotated_norm + cy;
        // populate image
        int x_round = round(x_prime[i]);
        int y_round = round(y_prime[i]);
        float gaussian;

        if (x_round >= 1 && x_round <= width && y_round >= 1 && y_round <= height)
        {
            float fx_div_z_rotated_ti = fx * z_rotated_inv * t_norm;
            float fy_div_z_rotated_ti = fy * z_rotated_inv * t_norm;
            float del_x_del_theta_y = fx_div_z_rotated_ti * (1 + x_unprojected[i] * x_rotated_norm);
            float del_x_del_theta_z = -fx_div_z_rotated_ti * y_unprojected[i];
            float del_x_del_theta_x = del_x_del_theta_z * x_rotated_norm;
            float del_y_del_theta_x = fy_div_z_rotated_ti * (-1 - y_unprojected[i] * y_rotated_norm);
            float del_y_del_theta_z = fy_div_z_rotated_ti * x_unprojected[i];
            float del_y_del_theta_y = del_y_del_theta_z * y_rotated_norm;

            for (int row = max(1, y_round - 2); row < min(height, y_round + 3); row++)
            {
                for (int col = max(1, x_round - 2); col < min(width, x_round + 3); col++)
                {
                    // TODO: make a LUT for the values here rounded to a certain s.f. and see if there is a speed-up
                    float x_diff = col - x_prime[i];
                    float y_diff = row - y_prime[i];
                    gaussian = exp((-x_diff * x_diff - y_diff * y_diff) / 2);
                    int idx = (row - 1) * (width) + col - 1;
                    atomicAdd(&image[idx], gaussian);
                    image_sum += gaussian;
                    float del_x = gaussian * (x_diff * del_x_del_theta_x + y_diff * del_y_del_theta_x);
                    atomicAdd(&image_del_x[idx], del_x);
                    image_sum_del_theta_x += del_x;
                    float del_y = gaussian * (x_diff * del_x_del_theta_y + y_diff * del_y_del_theta_y);
                    atomicAdd(&image_del_y[idx], del_y);
                    image_sum_del_theta_y += del_y;
                    float del_z = gaussian * (x_diff * del_x_del_theta_z + y_diff * del_y_del_theta_z);
                    atomicAdd(&image_del_z[idx], del_z);
                    image_sum_del_theta_z += del_z;
                }
            }
        }
    }
    float *sdata = SharedMemory<float>();
    uint16_t tid = threadIdx.x;

    // do reduction in shared mem

    // sum up to 128 elements

    float temp_sum;
    // image_sum
    sdata[tid] = image_sum;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum = image_sum + sdata[tid + 256];
    __syncthreads();
    // store contrast in 0 to 127
    if (tid < 128)
        temp_sum = image_sum + sdata[tid + 128];
    __syncthreads();
    // image_sum_del_theta_x
    sdata[tid] = image_sum_del_theta_x;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 128];
    __syncthreads();
    // store x in 128 to 255
    if (tid >= 128 && tid < 256)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_sum_del_theta_y
    sdata[tid] = image_sum_del_theta_y;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 128];
    __syncthreads();
    // store y in 256 to 383
    if (tid >= 256 && tid < 384)
    {
        temp_sum = sdata[tid - 256];
    }
    __syncthreads();
    // image_sum_del_theta_z
    sdata[tid] = image_sum_del_theta_z;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
    {
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 128];
    }
    __syncthreads();
    // store z in 384 to 512
    if (tid >= 384)
    {
        temp_sum = sdata[tid - 384];
    }
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x7F) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid & 0x7F) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    __syncthreads();

    if (tid == 0)
    {
        // image_sum
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_sum_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 256)
    {
        // image_sum_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 384)
    {
        // image_sum_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}

__global__ void warpEvents_(float fx, float fy, float cx, float cy, int height, int width, int num_events, const float *x_unprojected, const float *y_unprojected, float *x_prime, float *y_prime, float *t, const float rotation_x, const float rotation_y, const float rotation_z, int x_offset, int y_offset)
{
    // size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    // if (i < num_events)
    for (size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x); i < num_events; i += num_threads_in_grid)
    {
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t[i];
        float theta_y_t = rotation_y * t[i];
        float theta_z_t = rotation_z * t[i];

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + cx;
        y_prime[i] = fy * y_rotated_norm + cy;
    }
}
void warpEvents(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, const float rotation_x, const float rotation_y, const float rotation_z, int x_offset, int y_offset)
{
    int blockSize = 512; // The launch configurator returned block size
    int gridSize;        // The actual grid size needed, based on input size
    gridSize = (num_events + blockSize - 1) / blockSize;
    int smemSize = blockSize * sizeof(float);
    warpEvents_<<<gridSize, blockSize, smemSize>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, rotation_x, rotation_y, rotation_z, x_offset, y_offset);
}
__global__ void fillImageBilinear_(float fx, float fy, float cx, float cy, int height, int width, int num_events, const float *x_unprojected, const float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum)
{

    float image_sum = 0;
    float image_sum_del_theta_x = 0;
    float image_sum_del_theta_y = 0;
    float image_sum_del_theta_z = 0;
    float *image_del_x = image + height * width;
    float *image_del_y = image + height * width * 2;
    float *image_del_z = image + height * width * 3;
    // size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    // if (i < num_events)
    for (size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x); i < num_events; i += num_threads_in_grid)
    {
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t[i];
        float theta_y_t = rotation_y * t[i];
        float theta_z_t = rotation_z * t[i];

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + cx;
        y_prime[i] = fy * y_rotated_norm + cy;
        // populate image

        // Bilinear
        int x_trunc = int(x_prime[i]);
        int y_trunc = int(y_prime[i]);
        if (x_trunc >= 1 && x_trunc <= width - 2 && y_trunc >= 1 && y_trunc <= height - 2)
        {

            // int idx1 = x_trunc - 1 + (y_trunc - 1) * width;
            // int idx2 = idx1 + 1;
            // int idx3 = idx1 + width;
            // int idx4 = idx3 + 1;

            int idx4 = x_trunc + y_trunc * width;
            int idx3 = idx4 - 1;
            int idx2 = idx4 - width;
            int idx1 = idx2 - 1;
            float x_diff = x_prime[i] - x_trunc;
            float y_diff = y_prime[i] - y_trunc;
            float del_x_del_theta_x, del_x_del_theta_y, del_x_del_theta_z, del_y_del_theta_x, del_y_del_theta_y, del_y_del_theta_z;
            float fx_div_z_rotated_ti = fx * z_rotated_inv * t[i];
            float fy_div_z_rotated_ti = fy * z_rotated_inv * t[i];
            del_x_del_theta_y = fx_div_z_rotated_ti * (1 + x_unprojected[i] * x_rotated_norm);
            del_x_del_theta_z = fx_div_z_rotated_ti * -y_unprojected[i];
            del_x_del_theta_x = del_x_del_theta_z * x_rotated_norm;
            del_y_del_theta_x = fy_div_z_rotated_ti * (-1 - y_unprojected[i] * y_rotated_norm);
            del_y_del_theta_z = fy_div_z_rotated_ti * x_unprojected[i];
            del_y_del_theta_y = del_y_del_theta_z * y_rotated_norm;
            // float d1x = -(1 - y_diff);
            // float d1y = -(1 - x_diff);
            float d2x = 1 - y_diff;
            float d2y = -x_diff;
            float d3x = -y_diff;
            float d3y = 1 - x_diff;
            float d4x = y_diff;
            float d4y = x_diff;

            float d1x = -d2x;
            float d1y = -d3y;

            // float im1 = (1 - x_diff) * (1 - y_diff);
            float im1 = d3y * d2x;
            // float im2 = (x_diff) * (1 - y_diff);
            float im2 = d4y * d2x;
            // float im3 = (1 - x_diff) * (y_diff);
            float im3 = d3y * y_diff;
            float im4 = (x_diff) * (y_diff);
            image_sum = im1 + im2 + im3 + im4;
            atomicAdd(&image[idx1], im1);
            atomicAdd(&image[idx2], im2);
            atomicAdd(&image[idx3], im3);
            atomicAdd(&image[idx4], im4);
            float dx1 = d1x * del_x_del_theta_x + d1y * del_y_del_theta_x;
            float dx2 = d2x * del_x_del_theta_x + d2y * del_y_del_theta_x;
            float dx3 = d3x * del_x_del_theta_x + d3y * del_y_del_theta_x;
            float dx4 = d4x * del_x_del_theta_x + d4y * del_y_del_theta_x;
            image_sum_del_theta_x = dx1 + dx2 + dx3 + dx4;

            atomicAdd(&image_del_x[idx1], dx1);
            atomicAdd(&image_del_x[idx2], dx2);
            atomicAdd(&image_del_x[idx3], dx3);
            atomicAdd(&image_del_x[idx4], dx4);
            float dy1 = d1x * del_x_del_theta_y + d1y * del_y_del_theta_y;
            float dy2 = d2x * del_x_del_theta_y + d2y * del_y_del_theta_y;
            float dy3 = d3x * del_x_del_theta_y + d3y * del_y_del_theta_y;
            float dy4 = d4x * del_x_del_theta_y + d4y * del_y_del_theta_y;
            image_sum_del_theta_y = dy1 + dy2 + dy3 + dy4;
            atomicAdd(&image_del_y[idx1], dy1);
            atomicAdd(&image_del_y[idx2], dy2);
            atomicAdd(&image_del_y[idx3], dy3);
            atomicAdd(&image_del_y[idx4], dy4);
            float dz1 = d1x * del_x_del_theta_z + d1y * del_y_del_theta_z;
            float dz2 = d2x * del_x_del_theta_z + d2y * del_y_del_theta_z;
            float dz3 = d3x * del_x_del_theta_z + d3y * del_y_del_theta_z;
            float dz4 = d4x * del_x_del_theta_z + d4y * del_y_del_theta_z;
            image_sum_del_theta_z = dz1 + dz2 + dz3 + dz4;
            atomicAdd(&image_del_z[idx1], dz1);
            atomicAdd(&image_del_z[idx2], dz2);
            atomicAdd(&image_del_z[idx3], dz3);
            atomicAdd(&image_del_z[idx4], dz4);
        }
    }
    float *sdata = SharedMemory<float>();
    uint16_t tid = threadIdx.x;

    // do reduction in shared mem

    // sum up to 128 elements

    float temp_sum;
    // image_sum
    sdata[tid] = image_sum;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum = image_sum + sdata[tid + 256];
    __syncthreads();
    // store contrast in 0 to 127
    if (tid < 128)
        temp_sum = image_sum + sdata[tid + 128];
    __syncthreads();
    // image_sum_del_theta_x
    sdata[tid] = image_sum_del_theta_x;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 128];
    __syncthreads();
    // store x in 128 to 255
    if (tid >= 128 && tid < 256)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_sum_del_theta_y
    sdata[tid] = image_sum_del_theta_y;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 128];
    __syncthreads();
    // store y in 256 to 383
    if (tid >= 256 && tid < 384)
    {
        temp_sum = sdata[tid - 256];
    }
    __syncthreads();
    // image_sum_del_theta_z
    sdata[tid] = image_sum_del_theta_z;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
    {
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 128];
    }
    __syncthreads();
    // store z in 384 to 512
    if (tid >= 384)
    {
        temp_sum = sdata[tid - 384];
    }
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x7F) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid & 0x7F) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    __syncthreads();

    if (tid == 0)
    {
        // image_sum
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_sum_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 256)
    {
        // image_sum_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 384)
    {
        // image_sum_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}
void fillImageBilinear(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum)
{
    // const int num_sm = 8; // Jetson Orin NX
    // const int blocks_per_sm = 4;
    // const int threads_per_block = 128;
    int blockSize = 512; // The launch configurator returned block size
    // int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
    //                                    fillImageBilinear_, 0, 0);
    // Round up according to array size
    gridSize = std::min(128, (num_events + blockSize - 1) / blockSize);

    int smemSize = blockSize * sizeof(float);
    fillImageBilinear_<<<gridSize, blockSize, smemSize>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, image, rotation_x, rotation_y, rotation_z, contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);
}
void fillImage(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, hipStream_t const *stream, int x_offset, int y_offset)
{
    // const int num_sm = 8; // Jetson Orin NX
    // const int blocks_per_sm = 4;
    // const int threads_per_block = 128;
    int blockSize = 512; // The launch configurator returned block size
    // int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
    //                                    fillImageBilinear_, 0, 0);
    // Round up according to array size
    gridSize = std::min(128, (num_events + blockSize - 1) / blockSize);

    int smemSize = blockSize * sizeof(float);
    fillImage_<<<gridSize, blockSize, smemSize, stream[0]>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, image, rotation_x, rotation_y, rotation_z, contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum, x_offset, y_offset);
}

__global__ void fillImageKronecker_(int height, int width, int num_events, float *x_prime, float *y_prime, int *image)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // populate image
        // check if coordinates are 3 pixels in of the boundary
        int x_round = round(x_prime[i]);
        int y_round = round(y_prime[i]);
        if (x_round >= 1 && x_round <= width && y_round >= 1 && y_round <= height)
        {
            int idx = (y_round - 1) * width + x_round - 1;
            atomicAdd(&image[idx], 1);
        }
    }
}
void fillImageKronecker(int height, int width, int num_events, float *x_prime, float *y_prime, int *image)
{
    const int num_sm = 8; // Jetson Orin NX
    const int blocks_per_sm = 4;
    const int threads_per_block = 128;
    // hipMemset(image, 0, height * width * sizeof(uint16_t));
    fillImageKronecker_<<<blocks_per_sm * num_sm, threads_per_block>>>(height, width, num_events, x_prime, y_prime, image);
}
int getMax(int *image, int height, int width)
{
    int *out;
    hipMalloc(&out, sizeof(int));
    size_t temp_cub_temp_size;
    int *temp_storage = NULL;
    hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, out, (height) * (width), hipcub::Max(), 0);
    hipDeviceSynchronize();
    hipMalloc(&temp_storage, temp_cub_temp_size);
    hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, out, (height) * (width), hipcub::Max(), 0);
    hipDeviceSynchronize();
    int maximum;
    hipMemcpy(&maximum, out, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(out);
    hipFree(temp_storage);
    return maximum;
}
__device__ volatile float mean_volatile[4] = {0};
// __global__ void getContrastDelBatchReduceHarder256_(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize, float *imagedebug)
__global__ void getContrastDelBatchReduceHarder256_(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize)
{
    cooperative_groups::grid_group grid = cooperative_groups::this_grid();
    float *image_del_x = image + num_elements;
    float *image_del_y = image + num_elements * 2;
    float *image_del_z = image + num_elements * 3;
    // START COPY
    float *sdata = SharedMemory<float>();
    float temp_sum = 0;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    if (tid < prev_gridsize)
    {

        if (blockIdx.x == 0)
        {
            temp_sum = contrast_block_sum[tid];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum = contrast_del_x_block_sum[tid];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum = contrast_del_y_block_sum[tid];
        }
        else if (blockIdx.x == 3)
        {
            temp_sum = contrast_del_z_block_sum[tid];
        }
    }
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        // write means
        if (blockIdx.x < 4)
        {
            mean_volatile[blockIdx.x] = temp_sum / num_elements;
        }
    }
    // __syncthreads();

    // END COPY
    float image_contrast = 0;
    float image_contrast_del_theta_x = 0;
    float image_contrast_del_theta_y = 0;
    float image_contrast_del_theta_z = 0;
    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t idx = thread_grid_idx;
    cooperative_groups::sync(grid);
    float mean = mean_volatile[0];
    float xmean = mean_volatile[1];
    float ymean = mean_volatile[2];
    float zmean = mean_volatile[3];

    while (idx < num_elements)
    {
        float image_norm = image[idx] - mean;
        float image_norm_x = image_del_x[idx] - xmean;
        float image_norm_y = image_del_y[idx] - ymean;
        float image_norm_z = image_del_z[idx] - zmean;
        image_contrast += image_norm * image_norm;
        image_contrast_del_theta_x += image_norm_x * image_norm;
        image_contrast_del_theta_y += image_norm_y * image_norm;
        image_contrast_del_theta_z += image_norm_z * image_norm;
        // imagedebug[idx] = image_norm * image_norm;
        // imagedebug[idx + num_elements] = image_norm_x * image_norm;
        // imagedebug[idx + num_elements * 2] = image_norm_y * image_norm;
        // imagedebug[idx + num_elements * 3] = image_norm_z * image_norm;
        idx += blockDim.x * gridDim.x;
    }
    // BEGIN DEBUG
    // sdata[tid] = 0;
    // if(blockIdx.x==0&&threadIdx.x==0){
    //     for(int idx_special=0;idx_special<num_elements;idx_special++)

    //     {
    //         float image_norm = image[idx_special] - mean;
    //         float image_norm_x = image_del_x[idx_special] - xmean;
    //         float image_norm_y = image_del_y[idx_special] - ymean;
    //         float image_norm_z = image_del_z[idx_special] - zmean;
    //         // image_contrast += image_norm * image_norm;
    //         // image_contrast_del_theta_x += image_norm_x * image_norm;
    //         // image_contrast_del_theta_y += image_norm_y * image_norm;
    //         // image_contrast_del_theta_z += image_norm_z * image_norm;
    //         // imagedebug[idx_special] = image_norm * image_norm;
    //         // imagedebug[idx_special + num_elements] = image_norm_x * image_norm;
    //         // imagedebug[idx_special + num_elements * 2] = image_norm_y * image_norm;
    //         // imagedebug[idx_special + num_elements * 3] = image_norm_z * image_norm;

    //         atomicAdd(&sdata[0], image_norm * image_norm);
    //         atomicAdd(&sdata[1], image_norm_x * image_norm);
    //         atomicAdd(&sdata[2], image_norm_y * image_norm);
    //         atomicAdd(&sdata[3], image_norm_z * image_norm);

    //     }

    //     contrast_block_sum[blockIdx.x] = sdata[0];
    //     contrast_del_x_block_sum[blockIdx.x] = sdata[1];
    //     contrast_del_y_block_sum[blockIdx.x] = sdata[2];
    //     contrast_del_z_block_sum[blockIdx.x] = sdata[3];
    //     // contrast_block_sum[blockIdx.x] = image_contrast;
    //     // contrast_del_x_block_sum[blockIdx.x] = image_contrast_del_theta_x;
    //     // contrast_del_y_block_sum[blockIdx.x] = image_contrast_del_theta_y;
    //     // contrast_del_z_block_sum[blockIdx.x] = image_contrast_del_theta_z;
    // }
    // else{

    //     contrast_block_sum[blockIdx.x] =0;
    //     contrast_del_x_block_sum[blockIdx.x] =0;
    //     contrast_del_y_block_sum[blockIdx.x] =0;
    //     contrast_del_z_block_sum[blockIdx.x] = 0;
    // }
    // return;
    // END DEBUG

    // do reduction in shared mem

    // sum up to 128 elements

    // float temp_sum;
    // image_contrast
    sdata[tid] = image_contrast;
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast = image_contrast + sdata[tid + 128];
    __syncthreads();
    // store contrast in 0 to 63
    if (tid < 64)
        temp_sum = image_contrast + sdata[tid + 64];
    __syncthreads();
    // image_contrast_del_theta_x
    sdata[tid] = image_contrast_del_theta_x;
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_x = image_contrast_del_theta_x + sdata[tid + 128];
    __syncthreads();
    if (tid < 64)
        sdata[tid] = image_contrast_del_theta_x = image_contrast_del_theta_x + sdata[tid + 64];
    __syncthreads();
    // store x in 64 to 127
    if (tid >= 64 && tid < 128)
    {
        temp_sum = sdata[tid - 64];
    }
    __syncthreads();
    // image_contrast_del_theta_y
    sdata[tid] = image_contrast_del_theta_y;
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_y = image_contrast_del_theta_y + sdata[tid + 128];
    __syncthreads();
    if (tid < 64)
        sdata[tid] = image_contrast_del_theta_y = image_contrast_del_theta_y + sdata[tid + 64];
    __syncthreads();
    // store y in 128 to 191
    if (tid >= 128 && tid < 192)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_contrast_del_theta_z
    sdata[tid] = image_contrast_del_theta_z;
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_z = image_contrast_del_theta_z + sdata[tid + 128];
    __syncthreads();
    if (tid < 64)
    {
        sdata[tid] = image_contrast_del_theta_z = image_contrast_del_theta_z + sdata[tid + 64];
    }
    __syncthreads();
    // store z in 192 to 255
    if (tid >= 192)
    {
        temp_sum = sdata[tid - 192];
    }
    __syncthreads();
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x3F) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }

    if (tid == 0)
    {
        // image_contrast
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 64)
    {
        // image_contrast_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_contrast_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 192)
    {
        // image_contrast_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}
__global__ void getContrastDelBatchReduceHarder_(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize)
{
    float *image_del_x = image + num_elements;
    float *image_del_y = image + num_elements * 2;
    float *image_del_z = image + num_elements * 3;
    // START COPY
    float *sdata = SharedMemory<float>();
    float temp_sum = 0;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    if (tid < prev_gridsize)
    {

        if (blockIdx.x == 0)
        {
            temp_sum = contrast_block_sum[tid];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum = contrast_del_x_block_sum[tid];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum = contrast_del_y_block_sum[tid];
        }
        else if (blockIdx.x == 3)
        {
            temp_sum = contrast_del_z_block_sum[tid];
        }
    }
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        if (blockIdx.x == 0)
        {
            means[0] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 1)
        {
            means[1] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 2)
        {
            means[2] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 3)
        {
            means[3] = temp_sum / num_elements;
        }
    }

    // END COPY
    float image_contrast = 0;
    float image_contrast_del_theta_x = 0;
    float image_contrast_del_theta_y = 0;
    float image_contrast_del_theta_z = 0;
    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    // size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    size_t idx = thread_grid_idx;
    // __syncthreads();
    auto g = cooperative_groups::this_grid();
    // auto g = cooperative_groups::this_thread_block();
    g.sync();
    float mean = means[0];
    float xmean = means[1];
    float ymean = means[2];
    float zmean = means[3];

    while (idx < num_elements)
    {
        float image_norm = image[idx] - mean;
        float image_norm_x = image_del_x[idx] - xmean;
        float image_norm_y = image_del_y[idx] - ymean;
        float image_norm_z = image_del_z[idx] - zmean;
        image_contrast += image_norm * image_norm;
        image_contrast_del_theta_x += image_norm_x * image_norm;
        image_contrast_del_theta_y += image_norm_y * image_norm;
        image_contrast_del_theta_z += image_norm_z * image_norm;
        idx += blockDim.x * gridDim.x;
    }

    // do reduction in shared mem

    // sum up to 128 elements

    // float temp_sum;
    // image_contrast
    sdata[tid] = image_contrast;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_contrast = image_contrast + sdata[tid + 256];
    __syncthreads();
    // store contrast in 0 to 127
    if (tid < 128)
        temp_sum = image_contrast + sdata[tid + 128];
    __syncthreads();
    // image_contrast_del_theta_x
    sdata[tid] = image_contrast_del_theta_x;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_contrast_del_theta_x = image_contrast_del_theta_x + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_x = image_contrast_del_theta_x + sdata[tid + 128];
    __syncthreads();
    // store x in 128 to 255
    if (tid >= 128 && tid < 256)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_contrast_del_theta_y
    sdata[tid] = image_contrast_del_theta_y;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_contrast_del_theta_y = image_contrast_del_theta_y + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_y = image_contrast_del_theta_y + sdata[tid + 128];
    __syncthreads();
    // store y in 256 to 383
    if (tid >= 256 && tid < 384)
    {
        temp_sum = sdata[tid - 256];
    }
    __syncthreads();
    // image_contrast_del_theta_z
    sdata[tid] = image_contrast_del_theta_z;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_contrast_del_theta_z = image_contrast_del_theta_z + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
    {
        sdata[tid] = image_contrast_del_theta_z = image_contrast_del_theta_z + sdata[tid + 128];
    }
    __syncthreads();
    // store z in 384 to 512
    if (tid >= 384)
    {
        temp_sum = sdata[tid - 384];
    }
    __syncthreads();
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x7F) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid & 0x7F) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }

    if (tid == 0)
    {
        // image_contrast
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_contrast_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 256)
    {
        // image_contrast_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 384)
    {
        // image_contrast_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}

// 4 blocks x threads
template <int prev_gridsize>
__global__ void getContrastDelBatchReduceHarderPt2_(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum)
{
    float *sdata = SharedMemory<float>();

    float temp_sum = 0;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    if (tid < prev_gridsize)
    {

        if (blockIdx.x == 0)
        {
            temp_sum = temp_sum = contrast_block_sum[tid];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum = contrast_del_x_block_sum[tid];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum = contrast_del_y_block_sum[tid];
        }
        else if (blockIdx.x == 3)
        {
            temp_sum = contrast_del_z_block_sum[tid];
        }
    }
    sdata[tid] = temp_sum;
    __syncthreads();

    if (prev_gridsize > 256 && (tid) < 256)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 256];
    }
    __syncthreads();
    if (prev_gridsize > 128 && (tid) < 128)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 128];
    }
    __syncthreads();
    if (prev_gridsize > 64 && (tid) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        if (blockIdx.x == 0)
        {
            contrast_block_sum[0] = temp_sum;
        }
        else if (blockIdx.x == 1)
        {
            contrast_block_sum[1] = temp_sum;
        }
        else if (blockIdx.x == 2)
        {
            contrast_block_sum[2] = temp_sum;
        }
        else
        {
            contrast_block_sum[3] = temp_sum;
        }
    }
}

// 4 blocks 128 threads
__global__ void meanPt2_(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int num_elements, float *means, int prev_gridsize)
{
    float *sdata = SharedMemory<float>();
    float temp_sum = 0;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    if (tid < prev_gridsize)
    {

        if (blockIdx.x == 0)
        {
            temp_sum = contrast_block_sum[tid];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum = contrast_del_x_block_sum[tid];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum = contrast_del_y_block_sum[tid];
        }
        else
        {
            temp_sum = contrast_del_z_block_sum[tid];
        }
    }
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid) < 32)
    {
        // warps of 32 threads are always in sync, no need to sync after this
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        if (blockIdx.x == 0)
        {
            means[0] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 1)
        {
            means[1] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 2)
        {
            means[2] = temp_sum / num_elements;
        }
        else
        {
            means[3] = temp_sum / num_elements;
        }
    }
}

void getContrastDelBatchReduce(float *image,
                               double *image_contrast, double *image_del_theta_contrast,
                               int height, int width,
                               float *contrast_block_sum,
                               float *contrast_del_x_block_sum,
                               float *contrast_del_y_block_sum,
                               float *contrast_del_z_block_sum,
                               float *means,
                               int num_events,
                               hipStream_t const *stream)
{
    // int dev = 0;
    // hipDeviceProp_t deviceProp;
    // hipGetDeviceProperties(&deviceProp, dev);
    // /// This will launch a grid that can maximally fill the GPU, on the default stream with kernel arguments
    // int numBlocksPerSm = 0;
    // // Number of threads my_kernel will be launched with
    // hipDeviceProp_t deviceProp;
    // hipGetDeviceProperties(&deviceProp, dev);
    // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, getContrastDelBatchReduceHarder_, 512, 0);
    // std::cout << numBlocksPerSm<<deviceProp.multiProcessorCount << std::endl;
    // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, getContrastDelBatchReduceHarder_, 256, 0);
    // std::cout << numBlocksPerSm<<deviceProp.multiProcessorCount << std::endl;
    // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, getContrastDelBatchReduceHarder_, 128, 0);
    // std::cout << numBlocksPerSm<<deviceProp.multiProcessorCount << std::endl;

    // int blockSize = 512; // The launch configurator returned block size
    int blockSize = 256; // The launch configurator returned block size

    int prev_blocksize = 512;
    // int prev_gridsize = (num_events + blockSize - 1) / blockSize;
    int prev_gridsize = std::min(128, (num_events + blockSize - 1) / prev_blocksize);

    // int gridSize = 85; // The actual grid size needed, based on input size
    // int gridSize = std::min(512, (height * width + blockSize - 1) / blockSize);
    // int gridSize = 5 * 8;
    int gridSize = 6 * 8;

    int smemSize = (blockSize <= 32) ? 2 * blockSize * sizeof(float) : blockSize * sizeof(float);

    // thrust::device_vector<float> imagedebug(height * width * 4);
    // float *imagedebugptr = thrust::raw_pointer_cast(imagedebug.data());
    // getContrastDelBatchReduceHarder_<<<gridSize, blockSize, smemSize>>>(image, height * width, means, contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum, prev_gridsize);
    int image_pixels = height * width;
    void *kernel_args[] = {
        (void *)&image,
        (void *)&image_pixels,
        (void *)&means,
        (void *)&contrast_block_sum,
        (void *)&contrast_del_x_block_sum,
        (void *)&contrast_del_y_block_sum,
        (void *)&contrast_del_z_block_sum,
        (void *)&prev_gridsize,
    };
    //    (void *)&imagedebugptr};
    // int dev = 0;
    // int supportsCoopLaunch = 0;
    // hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);
    // std::cout << supportsCoopLaunch << std::endl;

    checkCudaErrors(hipPeekAtLastError());
    hipLaunchCooperativeKernel((void *)getContrastDelBatchReduceHarder256_, gridSize, blockSize, kernel_args, smemSize, stream[0]);
    checkCudaErrors(hipPeekAtLastError());

    // START OF DEBUG
    // std::cout << "harder1 sum " << -thrustSum(contrast_block_sum, gridSize) / (height * width)
    //           << " " << -2 * thrustSum(contrast_del_x_block_sum, gridSize) / (height * width)
    //           << " " << -2 * thrustSum(contrast_del_y_block_sum, gridSize) / (height * width)
    //           << " " << -2 * thrustSum(contrast_del_z_block_sum, gridSize) / (height * width)
    //           << std::endl;

    // END DEBUG
    checkCudaErrors(hipPeekAtLastError());
    // checkCudaErrors(hipDeviceSynchronize());
    // checkCudaErrors(hipDeviceSynchronize());
    // std::cout << "thrust hardreduce             " << -thrust::reduce(imagedebug.begin(), imagedebug.begin() + height * width, 0.0, thrust::plus<float>()) / (height * width) << " "
    //           << -2 * thrust::reduce(imagedebug.begin() + height * width, imagedebug.begin() + height * width * 2, 0.0, thrust::plus<float>()) / (height * width) << " "
    //           << -2 * thrust::reduce(imagedebug.begin() + height * width * 2, imagedebug.begin() + height * width * 3, 0.0, thrust::plus<float>()) / (height * width) << " "
    //           << -2 * thrust::reduce(imagedebug.begin() + height * width * 3, imagedebug.begin() + height * width * 4, 0.0, thrust::plus<float>()) / (height * width) << " "
    //           << std::endl;
    if (height == 180 && width == 240)
        getContrastDelBatchReduceHarderPt2_<8 * 6><<<4, 128, 128 * sizeof(float), stream[0]>>>(contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);
    else if (height == 480 && width == 640)
        getContrastDelBatchReduceHarderPt2_<8 * 6><<<4, 512, 512 * sizeof(float), stream[0]>>>(contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);
    else if (height == 400 && width == 400)
        getContrastDelBatchReduceHarderPt2_<8 * 6><<<4, 512, 512 * sizeof(float), stream[0]>>>(contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);
    else if (height == 720 && width == 1280)
        getContrastDelBatchReduceHarderPt2_<8 * 6><<<4, 512, 512 * sizeof(float), stream[0]>>>(contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);
    else
    {
        getContrastDelBatchReduceHarderPt2_<8 * 6><<<4, 512, 512 * sizeof(float), stream[0]>>>(contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);
    }

    // checkCudaErrors(hipPeekAtLastError());
    hipMemsetAsync(image, 0, (height) * (width) * sizeof(float) * 4, stream[1]);
    // checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipStreamSynchronize(stream[0]));
    checkCudaErrors(hipStreamSynchronize(stream[1]));
    // hipDeviceSynchronize();
    {
        // nvtx3::scoped_range r{"final contrast"};
        int num_el = height * width;
        image_contrast[0] = -contrast_block_sum[0] / num_el;
        image_del_theta_contrast[0] = -2 * contrast_block_sum[1] / num_el;
        image_del_theta_contrast[1] = -2 * contrast_block_sum[2] / num_el;
        image_del_theta_contrast[2] = -2 * contrast_block_sum[3] / num_el;
    }
}

__device__ float getRandom(uint64_t seed, int tid, int threadCallCount)
{
    hiprandState s;
    hiprand_init(seed + tid + threadCallCount, 0, 0, &s);
    // return hiprand_uniform(&s);
    return hiprand_log_normal(&s, 1e-16, 10.0);
}
__global__ void one_step_kernel_(uint64_t seed, float *randoms, int numel)
{
    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    for (size_t idx = thread_grid_idx; idx < numel; idx += num_threads_in_grid)
    {

        randoms[idx] = getRandom(seed, idx, 0);
    }
}

void one_step_kernel(uint64_t seed, float *randoms, int numel)
{
    one_step_kernel_<<<43, 1024>>>(seed, randoms, numel);
}
__global__ void rescaleIntensity_(int *image, uint8_t *output_image, float maximum, int numel)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t idx = thread_grid_idx; idx < numel; idx += num_threads_in_grid)
    {
        output_image[idx] = (uint8_t)min(255, max(0, (int)(255 * image[idx] / (maximum / 2))));
    }
}
void rescaleIntensity(int *image, uint8_t *output_image, float maximum, int height, int width)
{
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
                     // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       rescaleIntensity_, 0, 0);
    // Round up according to array size
    int numel = height * width;
    gridSize = (numel + blockSize - 1) / blockSize;
    rescaleIntensity_<<<gridSize, blockSize>>>(image, output_image, maximum, numel);
    hipDeviceSynchronize();
}